#include "hip/hip_runtime.h"
#include "../include/CGemmWithC.h"
#include "../include/common.h"
#include "../include/mycuda.h"
#include <cstdio>
#include <cstdlib>

using namespace std;

void checkResult(float *hostRef, float *gpuRef, const int N) {
  double epsilon = 1.0E-8;
  bool match = 1;

  for (int i = 0; i < N; i++) {
    if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
      match = 0;
      printf("Arrays do not match!\n");
      printf("%d\n", i);
      printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
      break;
    }
  }

  if (match)
    printf("Arrays match.\n\n");

  return;
}

void initialData(float *ip, int size) {
  // generate different seed for random number
  time_t t;
  srand((unsigned)time(&t));

  for (int i = 0; i < size; i++) {
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  }

  return;
}

void printMatrix(float *matrix, int size, int nx, int ny) {
  printf("Matrix: \n");
  float *A = matrix;
  for (int i = 0; i < size; i++) {

    printf("%f ", *(A + i));
    if ((i + 1) % ny == 0) {
      printf("\n");
    }
  }
  printf("\n\n");
}

__global__ void MulMatrixOnDevice(int M, int N, int K, float alpha, float *A,
                                  float *B, float beta, float *C) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  // printf("%f %f\n", A[row * N + k] , B[k * N + col])
  if (row < M && col < N) {
    float temp = 0.0;
    for (int k = 0; k < K; k++) {
      temp += A[row * N + k] * B[k * N + col];
      // printf("%f %f\n", A[row * N + k], B[k * N + col]);
    }
    C[row * N + col] = alpha * temp + beta * C[row * N + col];
  }
}

int main(int argc, char **argv) {
  float *hostA;
  float *hostB;
  float *hostC;
  float *hostRef;
  float *gpuRef;

  int nx = 1280;
  int ny = 1280;
  int elemNum = nx * ny;

  // 给主机上的三个矩阵分配内存
  hostA = (float *)malloc(elemNum * sizeof(float));
  hostB = (float *)malloc(elemNum * sizeof(float));
  hostC = (float *)malloc(elemNum * sizeof(float));
  hostRef = (float *)malloc(elemNum * sizeof(float));
  gpuRef = (float *)malloc(elemNum * sizeof(float));

  // 主机上的三个矩阵初始化数据
  initialData(hostA, elemNum);
  initialData(hostB, elemNum);
  initialData(hostC, elemNum);
  memset(hostRef, 0, elemNum * sizeof(float));
  memset(gpuRef, 0, elemNum * sizeof(float));

  // 测试主机上的三个矩阵是否已经被初始化数据
  // printMatrix(hostA, elemNum, nx, ny);
  // printMatrix(hostB, elemNum, nx, ny);
  // printMatrix(hostC, elemNum, nx, ny);

  double iStart, iElaps;

  // 在主机上执行矩阵乘法
  CGemmWithC girl;
  float alpha = 1.0;
  float beta = 1.0;
  iStart = seconds();
  girl.solveProblem(nx, nx, nx, alpha, hostA, hostB, beta, hostC, hostRef);
  iElaps = seconds();
  // girl.print(hostRef, elemNum); // 测试输出hostdef
  printf("MulMatrixOnHost Time elapsed %f sec\n", iElaps - iStart);

  // 使用cuda kernel 来执行矩阵乘法
  dim3 blockDim(elemNum / 8, elemNum / 8);
  dim3 gridDim(8, 8);

  // dim3 blockDim(16, 16);
  // dim3 gridDim((ny + blockDim.x - 1) / blockDim.x,
  //              (nx + blockDim.y - 1) / blockDim.y);

  float *deviceA;
  float *deviceB;
  float *deviceC;
  CHECK(hipMalloc((float **)&deviceA, elemNum * sizeof(float)));
  CHECK(hipMalloc((float **)&deviceB, elemNum * sizeof(float)));
  CHECK(hipMalloc((float **)&deviceC, elemNum * sizeof(float)));
  CHECK(hipMemcpy(deviceA, hostA, elemNum * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(deviceB, hostB, elemNum * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(deviceC, hostC, elemNum * sizeof(float),
                   hipMemcpyHostToDevice));
  iStart = seconds();
  MulMatrixOnDevice<<<gridDim, blockDim>>>(nx, nx, nx, alpha, deviceA, deviceB,
                                           beta, deviceC);
  iElaps = seconds();
  // girl.print(gpuRef, elemNum);
  printf("MulMatrixOnDevice Time elapsed %f sec\n", iElaps - iStart);
  CHECK(hipMemcpy(gpuRef, deviceC, elemNum * sizeof(float),
                   hipMemcpyDeviceToHost));
  // girl.print(hostRef, elemNum);
  // girl.print(gpuRef, elemNum);
  checkResult(hostRef, gpuRef, elemNum);
  CHECK(hipDeviceSynchronize());
  // 使用cublas 执行矩阵乘法
  return 0;
}
