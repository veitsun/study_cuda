#include "hip/hip_runtime.h"
#include "../include/CGemmWithC.h"
#include "../include/common.h"
#include "../include/mycuda.h"
#include <cstdio>
#include <cstdlib>

using namespace std;

void checkResult(float *hostRef, float *gpuRef, const int N) {
  double epsilon = 1.0E-8;
  bool match = 1;

  for (int i = 0; i < N; i++) {
    if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
      match = 0;
      printf("Arrays do not match!\n");
      printf("%d\n", i);
      printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
      break;
    }
  }

  if (match)
    printf("Arrays match.\n\n");

  return;
}

void initialData(float *ip, int size) {
  // generate different seed for random number
  time_t t;
  srand((unsigned)time(&t));

  for (int i = 0; i < size; i++) {
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  }

  return;
}

void printMatrix(float *matrix, int size, int nx, int ny) {
  printf("Matrix: \n");
  float *A = matrix;
  for (int i = 0; i < size; i++) {

    printf("%f ", *(A + i));
    if ((i + 1) % ny == 0) {
      printf("\n");
    }
  }
  printf("\n\n");
}

// __global__ void MulMatrixOnDevice(float *A, float *B, float *C, int nx,
//                                   int ny) {}

__global__ void MulMatrixOnDevice(int M, int N, int K, float alpha, float *A,
                                  float *B, float beta, float *C) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < M && col < N) {
    float temp = 0.0;
    for (int k = 0; k < K; k++) {
      temp += A[row * K + k] * B[k * N + col];
    }
    C[row * N + col] = alpha * temp + beta * C[row * N + col];
  }
}

int main(int argc, char **argv) {
  float *hostA;
  float *hostB;
  float *hostC;
  float *hostRef;
  float *gpuRef;

  int nx = 8;
  int ny = 8;
  int elemNum = nx * ny;

  // 给主机上的三个矩阵分配内存
  hostA = (float *)malloc(elemNum * sizeof(float));
  hostB = (float *)malloc(elemNum * sizeof(float));
  hostC = (float *)malloc(elemNum * sizeof(float));
  hostRef = (float *)malloc(elemNum * sizeof(float));
  gpuRef = (float *)malloc(elemNum * sizeof(float));

  // 主机上的三个矩阵初始化数据
  initialData(hostA, elemNum);
  initialData(hostB, elemNum);
  initialData(hostC, elemNum);
  memset(hostRef, 0, elemNum * sizeof(float));
  memset(gpuRef, 0, elemNum * sizeof(float));

  // 测试主机上的三个矩阵是否已经被初始化数据
  printMatrix(hostA, elemNum, nx, ny);
  printMatrix(hostB, elemNum, nx, ny);
  printMatrix(hostC, elemNum, nx, ny);

  double iStart, iElaps;

  // 在主机上执行矩阵乘法
  CGemmWithC girl;
  float alpha = 1.0;
  float beta = 1.0;
  iStart = seconds();
  girl.solveProblem(nx, nx, nx, alpha, hostA, hostB, beta, hostC, hostRef);
  iElaps = seconds();
  printf("MulMatrixOnHost Time elapsed %f sec\n", iElaps - iStart);

  // 使用cuda kernel 来执行矩阵乘法
  int blockSize = 64;
  dim3 block(blockSize);
  dim3 grid((elemNum + block.x - 1) / block.x);

  float *deviceA;
  float *deviceB;
  float *deviceC;
  CHECK(hipMalloc((float **)&deviceA, elemNum * sizeof(float)));
  CHECK(hipMalloc((float **)&deviceB, elemNum * sizeof(float)));
  CHECK(hipMalloc((float **)&deviceC, elemNum * sizeof(float)));
  CHECK(hipMemcpy(deviceA, hostA, elemNum * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(deviceB, hostB, elemNum * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(deviceC, hostC, elemNum * sizeof(float),
                   hipMemcpyHostToDevice));
  iStart = seconds();
  MulMatrixOnDevice<<<block, grid>>>(nx, nx, nx, alpha, deviceA, deviceB, beta,
                                     deviceC);
  iElaps = seconds();
  printf("MulMatrixOnDevice Time elapsed %f sec\n", iElaps - iStart);
  CHECK(hipMemcpy(gpuRef, deviceC, elemNum * sizeof(float),
                   hipMemcpyDeviceToHost));

  checkResult(hostRef, gpuRef, elemNum);

  // 使用cublas 执行矩阵乘法
  return 0;
}
